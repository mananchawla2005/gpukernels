#include <hip/hip_runtime.h>
__global__ void fused_add_relu_kernel(float* __restrict__ output, const float* __restrict__ input1, const float* __restrict__ input2, const int n){
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<n){
        float sum = input1[idx]+input2[idx];
        output[idx] = sum > 0 ? sum : 0;
    }
}

extern "C" void fused_add_relu(float* output_h, float* input1_h, float* input2_h, const int n){
    float* output_d, *input1_d, *input2_d;
    int size = n * sizeof(float);
    hipMalloc((void**)&output_d, size);
    hipMalloc((void**)&input1_d, size);
    hipMalloc((void**)&input2_d, size);
    hipMemcpy(input1_d, input1_h, size, hipMemcpyHostToDevice);
    hipMemcpy(input2_d, input2_h, size, hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    fused_add_relu_kernel<<<gridSize, blockSize>>>(output_d, input1_d, input2_d, n);
    hipDeviceSynchronize();
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);
    hipFree(output_d);
    hipFree(input1_d);
    hipFree(input2_d);
}