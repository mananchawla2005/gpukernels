#include <math.h>
#include <hip/hip_runtime.h>
#define PI 3.1415926535
__device__ float gaussian(int x, int y, float sigma){
    return exp(-(x*x+y*y)/(2.0f*sigma*sigma)) / (2.0f*PI*sigma*sigma);
}

__global__ void gaussian_blur_kernel(unsigned char* Pin_d, unsigned char* Pout_d, int width, int height, int stride) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int column = blockIdx.x*blockDim.x + threadIdx.x;
    if(row<height && column<width) {
        float sigma = stride/3.0f; // A convention
        int center_elem = stride/2;
        float blur_pixel[3] = {0.0f, 0.0f, 0.0f};
        float weight_sum = 0.0f;

        for (int i = row-center_elem; i <= row+center_elem; i++)
        {
            for (int j = column-center_elem; j <= column+center_elem; j++)
            {
                if(i>=0 && j>=0 && i < height && j < width) {
                    float weight = gaussian(i-row, j-column, sigma);
                    int pixel_idx = (i * width + j) * 3;
                    blur_pixel[0] += Pin_d[pixel_idx] * weight;
                    blur_pixel[1] += Pin_d[pixel_idx+1] * weight;
                    blur_pixel[2] += Pin_d[pixel_idx+2] * weight;
                    weight_sum += weight;
                }
            }
        }
        int out_idx = (row*width+column)*3;
        Pout_d[out_idx] = (unsigned char)(blur_pixel[0] / weight_sum);
        Pout_d[out_idx+1] = (unsigned char)(blur_pixel[1] / weight_sum);
        Pout_d[out_idx+2] = (unsigned char)(blur_pixel[2] / weight_sum);
    }
}

extern "C" void gaussian_blur(unsigned char* Pin_h, unsigned char* Pout_h, int width, int height, int stride){
    unsigned char* Pin_d, *Pout_d;
    const int image_size = width*height*3;
    hipMalloc((void**)&Pout_d, image_size);
    hipMalloc((void**)&Pin_d, image_size);
    hipMemcpy(Pin_d, Pin_h, image_size, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize(ceil(width/16.0), ceil(height/16.0));
    gaussian_blur_kernel<<<gridSize, blockSize>>>(Pin_d, Pout_d, width, height, stride);
    hipMemcpy(Pout_h, Pout_d, image_size, hipMemcpyDeviceToHost);
    hipFree(Pin_d);
    hipFree(Pout_d);
}