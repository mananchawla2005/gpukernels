#include <hip/hip_runtime.h>
#include <math.h>

__global__ void layer_norm_kernel(float* input, float* output, int rows, int cols){
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    if(row<rows) {
        extern __shared__ float SM[];
        float* shared_data = SM;

        for (size_t i = 0; i < cols; i++)
        {
            shared_data[i] = input[row*cols+i];
        }
        
        __syncthreads();

        float mean = 0.0f;
        for (size_t i = 0; i < cols; i++)
        {
            mean+=shared_data[i];
        }
        mean /= cols;

        float variance = 0.0f;
        for (int i = 0; i < cols; i++)
        {
            float diff = shared_data[i] - mean;
            variance += diff*diff;
        }
        variance/=cols;

        float inv_std = rsqrtf(variance+1e-5f);

        for (size_t i = 0; i < cols; i++)
        {
            output[row*cols+i] = (shared_data[i]-mean) * inv_std; 
        }
        
        
        
    }
    
}   


extern "C" void layer_norm(float* input_h, float* output_h, int rows, int cols) {
    float *input_d, *output_d;
    int size = rows*cols*sizeof(float);
    hipMalloc((void**)&input_d, size);
    hipMalloc((void**)&output_d, size);
    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = ceil(rows/float(blockSize));
    size_t shared_mem_size = cols * sizeof(float);
    layer_norm_kernel<<<gridSize, blockSize, shared_mem_size>>>(input_d, output_d, rows, cols);
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);
    hipFree(input_d);
    hipFree(output_d);

}