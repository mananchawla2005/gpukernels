#include <hip/hip_runtime.h>
#include <math.h>

__global__ void softmax_kernel(float* input, float* output, int rows, int cols) {
    int row = blockIdx.x;
    int tid = threadIdx.x;
    
    if (row < rows) {
        float local_max = -INFINITY;
        for (int i = tid; i < cols; i += blockDim.x) {
            if(input[row * cols + i]>local_max){
                local_max = input[row*cols+i];
            }
        }
        __shared__ float temp_max[256];  // Assuming max block size of 256
        temp_max[tid] = local_max;
        __syncthreads();
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (tid < stride) {
                temp_max[tid] = max(temp_max[tid], temp_max[tid + stride]);
            }
            // sync before next iteration
            __syncthreads();
        }
        
        float global_max = temp_max[0];
        __syncthreads();

        float local_exp_sum = 0.0f;
        for (int i = tid; i < cols; i += blockDim.x) {
            float exp_val = expf(input[row * cols + i] - global_max);
            local_exp_sum += exp_val;
            output[row * cols + i] = exp_val;  // Store intermediate results
        }
        
        __shared__ float total_sum;
        if (tid == 0) total_sum = 0.0f;
        __syncthreads();
        
        atomicAdd(&total_sum, local_exp_sum);
        __syncthreads();
       
        for (int i = tid; i < cols; i += blockDim.x) {
            output[row * cols + i] /= total_sum;
        }
    }
}


extern "C" void softmax(float* input_h, float* output_h, int rows, int cols) {
    float *input_d, *output_d;
    int size = rows * cols * sizeof(float);
    hipMalloc((void**)&input_d, size);
    hipMalloc((void**)&output_d, size);
    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = rows;
    
    softmax_kernel<<<gridSize, blockSize>>>(input_d, output_d, rows, cols);
    
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);
    hipFree(input_d);
    hipFree(output_d);
}