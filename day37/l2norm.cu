#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void l2NormAtomicKernel(const float* input, float* output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(output, input[i] * input[i]);
    }
}

float l2NormAtomic(const float* input_h, int n) {
    float *input_d, *output_d;
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    hipMalloc(&input_d, n * sizeof(float));
    hipMalloc(&output_d, sizeof(float));
    hipMemset(output_d, 0, sizeof(float));
    hipMemcpy(input_d, input_h, n * sizeof(float), hipMemcpyHostToDevice);
    l2NormAtomicKernel<<<numBlocks, blockSize>>>(input_d, output_d, n);
    hipDeviceSynchronize();
    float sum;
    hipMemcpy(&sum, output_d, sizeof(float), hipMemcpyDeviceToHost);
    float norm = sqrt(sum);
    hipFree(input_d);
    hipFree(output_d);
    
    return norm;
}

int main() {
    const int N = 1000000;
    float* input_h = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        input_h[i] = 1.0f;
    }
    float norm = l2NormAtomic(input_h, N);
    printf("L2 norm of vector using atomic: %f\n", norm);
    printf("Expected value: %f\n", sqrt(N));
    free(input_h);
    return 0;
}