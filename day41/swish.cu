#include <stdio.h>
#include <hip/hip_runtime.h>

// SWISH activation function: f(x) = x * sigmoid(x)
__global__ void swishKernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        float x = input[idx];
        float sigmoid_x = 1.0f / (1.0f + expf(-x));
        output[idx] = x * sigmoid_x;
    }
}

void swishActivation(float* input_d, float* output_d, int size) {
    const int blockSize = 256;
    const int numBlocks = (size + blockSize - 1) / blockSize;
    
    swishKernel<<<numBlocks, blockSize>>>(input_d, output_d, size);
}

int main() {
    const int size = 10;
    float input_h[size];
    float output_h[size];
    
    for (int i = 0; i < size; i++) {
        input_h[i] = i - 5.0f; 
    }
    
    float *input_d, *output_d;
    hipMalloc(&input_d, size * sizeof(float));
    hipMalloc(&output_d, size * sizeof(float));
    hipMemcpy(input_d, input_h, size * sizeof(float), hipMemcpyHostToDevice);
    
    swishActivation(input_d, output_d, size);
    
    hipMemcpy(output_h, output_d, size * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("SWISH Activation Function Results:\n");
    printf("--------------------------------\n");
    printf("   x    |  SWISH(x)  \n");
    printf("--------------------------------\n");
    
    for (int i = 0; i < size; i++) {
        printf(" %5.2f  |  %8.5f\n", input_h[i], output_h[i]);
    }
    
    hipFree(input_d);
    hipFree(output_d);
    
    return 0;
}