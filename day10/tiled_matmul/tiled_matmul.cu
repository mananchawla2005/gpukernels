#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define TILE_WIDTH 32

__global__ void tiled_matmul_kernel(float* M, float* N, float* P, int width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by*TILE_WIDTH+ty;
    int col = bx*TILE_WIDTH+tx;

    float Pval = 0;
    for (int phase = 0; phase < ceil(width/float(TILE_WIDTH)); phase++)
    {
        int m_col = phase * TILE_WIDTH + tx;
        int n_row = phase * TILE_WIDTH + ty;

        if (row < width && m_col < width)
            Mds[ty][tx] = M[row * width + m_col];
        else
            Mds[ty][tx] = 0.0f;
        
        if (n_row < width && col < width)
            Nds[ty][tx] = N[n_row * width + col];
        else
            Nds[ty][tx] = 0.0f;
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++)
        {
            Pval += Mds[ty][i]*Nds[i][tx];
        }
        __syncthreads();
    }
    if (row < width && col < width)
        P[row * width + col] = Pval;
    
    
}

extern "C" void tiled_matmul(float* M_h, float* N_h, float* P_h, int width) {
    float* M_d, *N_d, *P_d;
    int size = width*width*sizeof(float);
    hipMalloc((void**)&M_d, size);
    hipMalloc((void**)&N_d, size);
    hipMalloc((void**)&P_d, size);
    hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize(ceil(width/float(TILE_WIDTH)), ceil(width/float(TILE_WIDTH)));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    tiled_matmul_kernel<<<gridSize, blockSize>>>(M_d, N_d, P_d, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
}