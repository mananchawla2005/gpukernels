#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mseKernel(const float* predictions_d, const float* targets_d, float* result_d, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float diff = predictions_d[idx] - targets_d[idx];
        atomicAdd(result_d, diff * diff);
    }
}

extern "C" void solve(const float* predictions_h, const float* targets_h, int N, float &mse) {
    float *predictions_d = nullptr;
    float *targets_d = nullptr;
    float *result_d = nullptr;
    float result_h = 0.0f;
    
    hipMalloc((void**)&predictions_d, N * sizeof(float));
    hipMalloc((void**)&targets_d, N * sizeof(float));
    hipMalloc((void**)&result_d, sizeof(float));
    
    hipMemcpy(predictions_d, predictions_h, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(targets_d, targets_h, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_d, &result_h, sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    mseKernel<<<numBlocks, blockSize>>>(predictions_d, targets_d, result_d, N);
    hipDeviceSynchronize();
    
    hipMemcpy(&result_h, result_d, sizeof(float), hipMemcpyDeviceToHost);
    mse = result_h / N;
    
    hipFree(predictions_d);
    hipFree(targets_d);
    hipFree(result_d);
}

int main() {
    const int N = 4;
    float predictions_h[N] = {1.0f, 2.0f, 3.0f, 4.0f};
    float targets_h[N] = {1.5f, 2.5f, 3.5f, 4.5f};
    float mse = 0.0f;
    
    solve(predictions_h, targets_h, N, mse);
    std::cout << "MSE: " << mse << std::endl;
    return 0;
}