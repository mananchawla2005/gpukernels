#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n){
        C[i]=A[i]+B[i];
    }
}

extern "C" void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    float *A_d, *B_d, *C_d;
    int size = n* sizeof(float);
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    // Error Checking
    hipError_t err = hipMalloc((void**)&C_d, size);
    if(err!=hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    
    int blocksPerGrid = (n +255 / 256);
    printf("%d", n);
    vecAddKernel<<<blocksPerGrid, 256>>>(A_d, B_d, C_d, n);
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
