#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define TILE_WIDTH 32
#define COARSE_FACTOR 8

__global__ void tiled_matmul_coarsened_kernel(float* M, float* N, float* P, int width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by*TILE_WIDTH+ty;
    int colStart = bx*TILE_WIDTH*COARSE_FACTOR+tx;

    float Pval[COARSE_FACTOR];
    for (size_t c = 0; c < COARSE_FACTOR; c++)
    {
        Pval[c] = 0.0f;
    }
    
    for (int phase = 0; phase < ceil(width/float(TILE_WIDTH)); phase++)
    {
        int m_col = phase * TILE_WIDTH + tx;
        
        if (row < width && m_col < width) {
            
            Mds[ty][tx] = M[row * width + m_col];
        }
        else {
            Mds[ty][tx] = 0.0f;
        }
        
        for (size_t c = 0; c < COARSE_FACTOR; c++)
        {
            int n_row = phase * TILE_WIDTH + ty;
            int col = colStart+c*TILE_WIDTH;
            if (n_row < width && col < width)
                Nds[ty][tx] = N[n_row * width + col];
            else
                Nds[ty][tx] = 0.0f;
            __syncthreads();
    
            for (int k = 0; k < TILE_WIDTH; k++)
            {
                Pval[c] += Mds[ty][k]*Nds[k][tx];
            }
            __syncthreads();
        }
    }
    for (size_t c = 0; c < COARSE_FACTOR; c++)
    {
        int col = colStart+c*TILE_WIDTH;
        if (row < width && col < width)
            P[row * width + col] = Pval[c];
    }
    
    
}

extern "C" void tiled_matmul_coarsened(float* M_h, float* N_h, float* P_h, int width) {
    float* M_d, *N_d, *P_d;
    int size = width*width*sizeof(float);
    hipMalloc((void**)&M_d, size);
    hipMalloc((void**)&N_d, size);
    hipMalloc((void**)&P_d, size);
    hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
    // dim3 blockSize(16, 16);
    // dim3 gridSize(ceil(width/16.0), ceil(width/16.0));
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((width + TILE_WIDTH * COARSE_FACTOR - 1) / (TILE_WIDTH * COARSE_FACTOR), 
                  (width + TILE_WIDTH - 1) / TILE_WIDTH);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    tiled_matmul_coarsened_kernel<<<gridSize, blockSize>>>(M_d, N_d, P_d, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
}