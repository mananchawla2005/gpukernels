#include <hip/hip_runtime.h>

__global__ void dynamic_tanh_kernel(float* input, float* output, int n, float* alpha, float* weight, float* bias){
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<n){
        float scaled_input = alpha[0] * input[idx];
        float ex = expf(scaled_input);
        float emx = expf(-scaled_input);
        float tanh_val = (ex-emx)/(ex+emx);
        output[idx] = tanh_val*weight[idx] + bias[idx];
    }
    
}

extern "C" void dynamic_tanh(float* input_h, float* weight_h, float* output_h, int n, float* alpha_h, float* bias_h){
    float* input_d, *output_d, *weight_d, *alpha_d, *bias_d;
    int size = n*sizeof(float);
    hipMalloc((void**)&input_d, size);
    hipMalloc((void**)&output_d, size);
    hipMalloc((void**)&weight_d, size);
    hipMalloc((void**)&bias_d, size);
    hipMalloc((void**)&alpha_d, sizeof(float));
    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);
    hipMemcpy(weight_d, weight_h, size, hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_d, bias_h, size, hipMemcpyHostToDevice);
    int blockSize = 16*16;
    int gridSize = ceil(n/float(blockSize));
    dynamic_tanh_kernel<<<gridSize, blockSize>>>(input_d, output_d, n, alpha_d, weight_d, bias_d);
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);
    hipFree(input_d);
    hipFree(output_d);
    hipFree(weight_d);
    hipFree(alpha_d);
    hipFree(bias_d);
}